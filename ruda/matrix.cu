#include <hipblas.h>
#include "ruda/matrix.h"

float* ruda_mm32(const float* a, const float* b, const int m, const int k, const int n) {
  size_t const a_bytes = m * k * sizeof(float);
  size_t const b_bytes = k * n * sizeof(float);
  size_t const c_bytes = m * n * sizeof(float);
  
  float* c = (float*)malloc(c_bytes);

  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, a_bytes);
  hipMalloc(&d_B, b_bytes);
  hipMalloc(&d_C, c_bytes);

  hipMemcpy(d_A, a, a_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, b, b_bytes, hipMemcpyHostToDevice);

  const float alf = 1;
  const float bet = 0;
  const float *alpha = &alf;
  const float *beta = &bet;

  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Do the actual multiplication
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A, m, d_B, k, beta, d_C, m);

  // Destroy the handle
  hipblasDestroy(handle);

  hipMemcpy(c, d_C, c_bytes, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return c;
}

